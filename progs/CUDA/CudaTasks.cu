#include "hip/hip_runtime.h"
#include "CudaTasks.h"

__global__ void kernel(unsigned char* data, int img_size, int B, int* out) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < B) {
    int index = (img_size * 3 - 3) - tid * 3;
    int ans = 0;

    for (index; index >= 0; index -= B * 3) {
      if (data[index] * data[index + 1] * data[index + 2] < 1000) {
        ans++;
      }
    }
    out[tid] = ans;
    printf("\tThread-%d. Number of pixels: %d \n", tid + 1, ans);
  }
  __syncthreads();
}


void CudaTask(int B, const BMPFile* bmpf) {
  unsigned char *d_data;
  int *d_out;

  int img_size = bmpf->dhdr.width * bmpf->dhdr.height;
  unsigned char* h_data = bmpf->data;
  int *out = (int*)malloc(sizeof(int) * B);
  
  for(int i = 0; i < B; i++) { out[i] = 0; }

  hipMalloc((unsigned char**)&d_data, sizeof(unsigned char) * img_size);
  hipMalloc((int**)&d_out, sizeof(int) * B);

  hipMemcpy(d_data, h_data, sizeof(unsigned char) * img_size, hipMemcpyHostToDevice);

  int block_size = 32;
  int grid_size = 1;
  kernel<<<grid_size, block_size>>>(d_data, img_size, B, d_out);

  hipMemcpy(out, d_out, sizeof(int) * B, hipMemcpyDeviceToHost);

  int res = 0;
  for(int i = 0; i < B; i++) {
    res += out[i];
  }
  printf("\tTotal number of pixels: %d\n\n", res);

  hipFree(d_data);
  hipFree(d_out);
  free(out);
}