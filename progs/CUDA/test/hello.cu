
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}


int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("%f ", out[i]);
    }

    // Cleanup after kernel execution
    hipFree(d_a); hipFree(d_b); hipFree(d_out);
    free(a); free(b); free(out);
}




// #include "cuda_runtime.h"
// #include <stdio.h>

// __global__ void cuda_hello(){
//     printf("Hello World from GPU!\n");
// }

// int main() {
//     cuda_hello<<<1,1>>>(); 
//     printf("It's okey\n");
//     return 0;
// }
